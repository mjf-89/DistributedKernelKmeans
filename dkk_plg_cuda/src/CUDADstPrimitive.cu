#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#include "CUDADstPrimitive.h"

__global__ void cuKernelDst(float *data, float *dst)
{
	extern __shared__ float s_dst[];

	int t_id = threadIdx.x;
	int dst_id = blockIdx.y * gridDim.x + blockIdx.x;
	int data_i = blockIdx.x * blockDim.x + t_id;
	int data_j = blockIdx.y * blockDim.x + t_id;
	
	//compute components distance on shared memory
	s_dst[t_id]= (data[data_i]-data[data_j])*(data[data_i]-data[data_j]);
	
	//binary-tree reduction of components distance
	__syncthreads();
	for(int i=1; i<blockDim.x; i*=2){
		if(t_id%(i*2)==0)
			s_dst[t_id] += s_dst[t_id+i];
		__syncthreads();
	}

	//output distance
	if(t_id==0) dst[dst_id] = s_dst[0];
}

namespace DKK{
void CUDADstPrimitive::init()
{
}

void CUDADstPrimitive::execute()
{
	dim3 blocks(data->rows(),data->rows(),1), threads(data->cols(),1,1);

	hipMalloc((void**)&d_data, data->length()*sizeof(float));
	hipMemcpy(d_data, data->bff(0), data->length()*sizeof(float),hipMemcpyHostToDevice);

	hipMalloc((void**)&d_dst, dst->length()*sizeof(float));

	cuKernelDst<<<blocks,threads,threads.x>>> (d_data, d_dst);

	hipMemcpy(dst->bff(0), d_dst, dst->length()*sizeof(float),hipMemcpyDeviceToHost);
}

CUDADstPrimitive::~CUDADstPrimitive()
{
	hipFree(d_data);
}

}

